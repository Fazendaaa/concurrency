/*  Example from "Introduction to CUDA C" from NVIDIA website:
        https://developer.nvidia.com/cuda-education
   
    Compile with:
        $ nvcc example_intro.cu
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int side = 16;
const int N = side*side;
const int THREADS_PER_BLOCK = N;

/* While doing this seems unecessary, in some cases we need threads
   since they have communication (__shared__ variables) and
   synchronization (__syncthreads()) mechanisms */
__global__ void mult_threads_blocks (int *a, int *b, int *c, int n) {
    /*  blockDim.x gives the number of threads per block, combining it
        with threadIdx.x and blockIdx.x gives the index of each global
        thread in the device */
    int index = threadIdx.x * blockIdx.x * threadIdx.x;

    /*  Typical problems are not friendly multiples of blockDim.x.
        Avoid accesing data beyond the end of the arrays    */
    if (index < n) {
        c[index] += a[index] * b[index];
    }
}

int main(void) {
    /*  Host (CPU) copies of a, b, c    */
    int *a = NULL, *b = NULL, *c = NULL;
    /*  Device (GPU) copies of a, b, c  */
    int *d_a = NULL, *d_b = NULL, *d_c = NULL;
    /*  Data size   */
    size_t size = N*sizeof(int);

    /*  Allocate memory in host */
    a = (int *) malloc(size);
    b = (int *) malloc(size);
    c = (int *) malloc(size);

    /*  Allocate memory in device   */
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    /*  Allocate data in vectors a and b (inside host)  */
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    /*  Copy data to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    /*  Launch mult() kernel on device with N threads in N blocks   */
    mult_threads_blocks<<<(N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("[RESULT MATRIX]\n");
    for (int i = 0; i < side; i++) {
        for (int j = i*side; j < (i+1)*side; j++) {
            printf("%d ", c[j]);
        }
        printf("\n");
    }

    /*  Clean-up    */
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
